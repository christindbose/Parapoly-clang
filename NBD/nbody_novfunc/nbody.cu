#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <chrono>

#define ALL __noinline__ __device__

#include "../configuration.h"

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort) exit(code);
    }
}

static const int kCudaBlockSize = 256;

class Body {
  public:
    float pos_x;
    float pos_y;
    float vel_x;
    float vel_y;
    float mass;
    float force_x;
    float force_y;
    ALL void initBody(int idx) {
        hiprandState rand_state;
        hiprand_init(kSeed, idx, 0, &rand_state);

        pos_x = 2 * hiprand_uniform(&rand_state) - 1;
        pos_y = 2 * hiprand_uniform(&rand_state) - 1;
        vel_x = (hiprand_uniform(&rand_state) - 0.5) / 1000;
        vel_y = (hiprand_uniform(&rand_state) - 0.5) / 1000;
        mass = (hiprand_uniform(&rand_state) / 2 + 0.5) * kMaxMass;
    }
    ALL Body(int idx) {
        // hiprandState rand_state;
        // hiprand_init(kSeed, idx, 0, &rand_state);

        // pos_x = 2 * hiprand_uniform(&rand_state) - 1;
        // pos_y = 2 * hiprand_uniform(&rand_state) - 1;
        // vel_x = (hiprand_uniform(&rand_state) - 0.5) / 1000;
        // vel_y = (hiprand_uniform(&rand_state) - 0.5) / 1000;
        // mass = (hiprand_uniform(&rand_state) / 2 + 0.5) * kMaxMass;
    }

    ALL float computeDistance(Body *other) {
        float dx;
        float dy;
        float dist;
        dx = this->pos_x - other->pos_x;
        dy = this->pos_y - other->pos_y;
        dist = sqrt(dx * dx + dy * dy);
        return dist;
    }
    ALL float computeForce(Body *other, float dist) {
        float F = kGravityConstant * this->mass * other->mass /
                  (dist * dist + kDampeningFactor);
        return F;
    }
    ALL void updateVelX() { this->vel_x += this->force_x * kDt / this->mass; }
    ALL void updateVelY() { this->vel_y += this->force_y * kDt / this->mass; }
    ALL void updatePosX() { this->pos_x += this->vel_x * kDt; }
    ALL void updatePosY() { this->pos_y += this->vel_y * kDt; }
    ALL void initForce() {
        this->force_x = 0;
        this->force_y = 0;
    }
    ALL void updateForceX(Body *other, float F) {
        float dx;
        float dy;
        float dist;
        dx = -1 * (this->pos_x - other->pos_x);
        dy = -1 * (this->pos_y - other->pos_y);
        dist = sqrt(dx * dx + dy * dy);
        this->force_x += F * dx / dist;
    }
    ALL void updateForceY(Body *other, float F) {
        float dx;
        float dy;
        float dist;
        dx = -1 * (this->pos_x - other->pos_x);
        dy = -1 * (this->pos_y - other->pos_y);
        dist = sqrt(dx * dx + dy * dy);
        this->force_y += F * dy / dist;
    }

    void add_checksum();

    // Only for rendering.
    ALL float pos_x_() const { return pos_x; }
    ALL float pos_y_() const { return pos_y; }
    ALL float mass_() const { return mass; }
};

__device__ float device_checksum;

__global__ void Body_compute_force(Body **dev_bodies) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    float dist;
    float F;
    if (id < kNumBodies) {
        dev_bodies[id]->initForce();
        // printf("%d ddd\n", id);
        for (int i = 0; i < kNumBodies; ++i) {
            // Do not compute force with the body itself.
            if (id != i) {
                dist = dev_bodies[id]->computeDistance(dev_bodies[i]);
                F = dev_bodies[id]->computeForce(dev_bodies[i], dist);
                dev_bodies[id]->updateForceX(dev_bodies[i], F);
                dev_bodies[id]->updateForceY(dev_bodies[i], F);
            }
        }
    }
}

__global__ void Body_update(Body **dev_bodies) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id < kNumBodies) {
        dev_bodies[id]->updateVelX();
        dev_bodies[id]->updateVelY();
        dev_bodies[id]->updatePosX();
        dev_bodies[id]->updatePosY();

        if (dev_bodies[id]->pos_x < -1 || dev_bodies[id]->pos_x > 1) {
            dev_bodies[id]->vel_x = -dev_bodies[id]->vel_x;
        }

        if (dev_bodies[id]->pos_y < -1 || dev_bodies[id]->pos_y > 1) {
            dev_bodies[id]->vel_y = -dev_bodies[id]->vel_y;
        }
    }

}

__device__ void Body_add_checksum(Body **dev_bodies, int id) {
    atomicAdd(&device_checksum,
              dev_bodies[id]->pos_x + dev_bodies[id]->pos_y * 2 +
                  dev_bodies[id]->vel_x * 3 + dev_bodies[id]->vel_y * 4);
}

__global__ void kernel_initialize_bodies(Body **bodies) {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < kNumBodies;
         i += blockDim.x * gridDim.x) {
        bodies[i] = new Body(/*idx*/ i);
        bodies[i]->initBody(i);
        
    }
}

__global__ void kernel_compute_force() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < kNumBodies;
         i += blockDim.x * gridDim.x) {
        // Body_compute_force(i);
    }
}

__global__ void kernel_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < kNumBodies;
         i += blockDim.x * gridDim.x) {
        // Body_update(i);
    }
}

__global__ void kernel_compute_checksum(Body **bodies) {
    device_checksum = 0.0f;
    for (int i = 0; i < kNumBodies; ++i) {
        Body_add_checksum(bodies, i);
    }
}

// void print_ptr_diff(Body **ptr) {
//     int i;
//     for (i = 1; i < kNumBodies / 100; i++) {
//       unsigned long long ptr2=(unsigned long long)ptr[i];
//       unsigned long long ptr1=(unsigned long long)ptr[i-1];
//         printf("[ptr[%d]-ptr[%d]]= %ull\n", i, i - 1,
//     (ptr2-  ptr1 ));
//     }
//   }
int main(int /*argc*/, char ** /*argv*/) {
    Body **dev_bodies;

    // Allocate and create Body objects.
    hipMallocManaged(&dev_bodies, sizeof(Body *) * kNumBodies);
    printf("init bodies...\n");
    kernel_initialize_bodies<<<128, 128>>>(dev_bodies);
    
    gpuErrchk(hipDeviceSynchronize());

    printf("init done...\n");
    auto time_start = std::chrono::system_clock::now();
    printf("Kernel exec...\n");
    for (int i = 0; i < kNumIterations; ++i) {
        if (i % 300 == 0) printf("Start: BodyComputeForce(%d)\n", i);
        Body_compute_force<<<(kNumBodies + kCudaBlockSize - 1) / kCudaBlockSize,
                             kCudaBlockSize>>>(dev_bodies);
        gpuErrchk(hipDeviceSynchronize());
        // printf("Body_compute_force(%d)\n",i);
        Body_update<<<(kNumBodies + kCudaBlockSize - 1) / kCudaBlockSize,
                      kCudaBlockSize>>>(dev_bodies);
        gpuErrchk(hipDeviceSynchronize());
        if (i % 300 == 0) printf("Finish: BodyComputeForce(%d)\n", i);
    }

    auto time_end = std::chrono::system_clock::now();
    auto elapsed = time_end - time_start;
    auto micros =
        std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

    printf("%lu\n", micros);

#ifndef NDEBUG
    kernel_compute_checksum<<<1, 1>>>(dev_bodies);
    gpuErrchk(hipDeviceSynchronize());

    float checksum;
    hipMemcpyFromSymbol(&checksum, HIP_SYMBOL(device_checksum), sizeof(device_checksum), 0,
                         hipMemcpyDeviceToHost);
    printf("Checksum: %f\n", checksum);
#endif  // NDEBUG

    hipFree(dev_bodies);

    return 0;
}
